#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <float.h>     /* provides DBL_EPSILON */
#include <sys/types.h>
#define FULL_MASK 0xffffffff

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

   
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__global__ void compute_diag_sub(double* dPtr,const unsigned long* atomsPtr,const unsigned long* fibersPtr,
                                 const double* valuesPtr,const double* DPtr,const unsigned long nFibers,const int nTheta,
                                 const unsigned long nCoeffs){   
    
    unsigned long k      =  threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long offset = 0;
    unsigned long stride = gridDim.x*blockDim.x;     
    while((k+offset)<nCoeffs){
        double val = 0;
        int atom_index = atomsPtr[k+offset];
        for (int i = 0; i < nTheta; i++){
            val += DPtr[atom_index+i]*DPtr[atom_index+i];
        }
        val = val*valuesPtr[k+offset]*valuesPtr[k+offset];
        atomicAdd(&dPtr[fibersPtr[k+offset]],val);
        offset+=stride;
    }   
    return;                
}

__global__ void M_times_w(
    double* YPtr,const unsigned long* atomsPtr,const unsigned long* voxelsPtr,
    const unsigned long* fibersPtr,const double* valuesPtr,const double* DPtr,
    const double* wPtr,const int nTheta,const unsigned long nVoxels,
    const unsigned long nCoeffs,const unsigned long* vox, const long nvox)
{  
    unsigned long long k =  (threadIdx.x/32) + (blockIdx.x*nc_mw) ;   
    if(k<nvox){
        unsigned long voxel_index  = voxelsPtr[vox[k]];
        __shared__ double y[nc_mw][Theta];
        int th_id = threadIdx.x%32;
        while(th_id<nTheta){
            y[threadIdx.x/32][th_id]=YPtr[voxel_index+th_id];
            th_id=th_id+32;
        }
        __syncwarp();
        #pragma unroll 8
        for(int t=vox[k];t<vox[k+1];t++){
            unsigned long fiber_index = fibersPtr[t]; 
            if(wPtr[fiber_index]){
                th_id=threadIdx.x%32;
                unsigned long atom_index  = atomsPtr[t];
                double val=wPtr[fiber_index]*valuesPtr[t];
                while(th_id<nTheta){
                    y[threadIdx.x/32][th_id]+= DPtr[atom_index+th_id]*val;
                    th_id=th_id+32;
                }
            }
            __syncwarp();
        }
        __syncwarp();
        th_id = threadIdx.x%32;
        while(th_id<nTheta){
            YPtr[voxel_index+th_id]=y[threadIdx.x/32][th_id];
            th_id=th_id+32;
        }
    }
    return;
}

__global__ void Mtransp_times_b(
    double* wPtr,const unsigned long* atomsPtr,const unsigned long* voxelsPtr,
    const unsigned long* fibersPtr,const double* valuesPtr,const double* DPtr,
    const double* YPtr,const unsigned long nFibers,const int nTheta,
    const long nCoeffs,const unsigned long* vox)
{  
    unsigned long long k  =  (threadIdx.x/32)+ (blockIdx.x*nc_my);     
        if(k<nCoeffs){
            unsigned long voxel_index  = voxelsPtr[k];
            unsigned long atom_index  = atomsPtr[k];
            double val;
            int th_id = threadIdx.x%32;
            while(th_id<nTheta){
                val = val + (DPtr[atom_index+th_id]*YPtr[voxel_index+th_id]);
                th_id=th_id+32;
            }
            __syncwarp();
            #pragma unroll 5
            for (int j = 16; j>=1; j=j/2){
                val+=__shfl_down_sync(FULL_MASK,val,j);
            }
            __syncwarp();
            if((threadIdx.x%32)==0){
                atomicAdd(&wPtr[fibersPtr[k]],val*valuesPtr[k]);
            }
        }
    return;
}